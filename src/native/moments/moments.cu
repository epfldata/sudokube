#include "hip/hip_runtime.h"
#include "moments.h"
#include "mycudaheaders.h"
#include <assert.h>
#include <chrono>


using namespace std::chrono;

inline
hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

//TODO Check if numMoments jstart jend need to be unsigned
__global__ void
computeKernel(int *moments, const int *data, const int numMoments, const int jstart,
               const int jend) {
    __shared__ int local[(ENTRIES_IN_BLOCK >> 1) * 33];
    __shared__ int tempvalues[ENTRIES_IN_BLOCK][
            VALUE_PER_MOMENT * 2 + 1]; // VALUES each of which can be at most 2 ints long
    int threadmoments[REC_SIZE_WORDS];
    const int blockMomentStart = blockIdx.x * ENTRIES_IN_BLOCK;
    const int yid = threadIdx.x >> 4; //REC_SIZE WORDS
    const int xid = threadIdx.x & 15;

    for (int i = yid;
         i < ENTRIES_IN_BLOCK && (blockMomentStart + i) < numMoments; i += (NUMCORES >> 4)) {
        local[(i << 4) + (i >> 1) + xid] = moments[(blockMomentStart + i) * REC_SIZE_WORDS + xid]; //add gap after 2 i
    }
    __syncthreads();
    int i = threadIdx.x / VALUE_PER_MOMENT; //multiple threads handle same moment
    int jy = threadIdx.x & (VALUE_PER_MOMENT - 1);
    for (int k = 0; k < KEY_SIZE_WORDS; k++) {
        threadmoments[k] = local[(i << 4) + (i >> 1) + k];
    }
    if (jy == 0) {
        *(T *) (threadmoments + KEY_SIZE_WORDS) = *(T *) (local + (i << 4) +(i >> 1) + KEY_SIZE_WORDS);
    } else
        *(T *) (threadmoments + KEY_SIZE_WORDS) = 0;
    __syncthreads();

    for (int joffset = jstart; joffset < jend; joffset += ENTRIES_IN_BLOCK) {
        //Copy DATA into shared memory
        for (int j = yid;
             j < ENTRIES_IN_BLOCK && (joffset + j) < jend; j += (NUMCORES >> 4)) {
            local[(j << 4) + (j >> 1) + xid] = data[(joffset + j) * REC_SIZE_WORDS + xid];
        }
        __syncthreads();
        if ((blockMomentStart + i) < numMoments) {
            for (int j = jy; j < ENTRIES_IN_BLOCK && (joffset + j) < jend; j += VALUE_PER_MOMENT) {
                int andRes = 1;
                for (int k = 0; k < KEY_SIZE_WORDS; k++) {
                    andRes &= ((threadmoments[k] & local[(j << 4) + (j >> 1) + k]) ==
                               threadmoments[k]);
                }
                *(T *) (threadmoments + KEY_SIZE_WORDS) += andRes * *(T *) (local + (j << 4) + (j >> 1) + KEY_SIZE_WORDS);
            }
        }
        __syncthreads();
    }
    if (i < ENTRIES_IN_BLOCK && (blockMomentStart + i) < numMoments) {
        //each jy can be atmost 2 ints
        *(T *) (tempvalues[i] + (jy << 1)) = *(T *) (threadmoments + KEY_SIZE_WORDS);
        __syncthreads();
        if (jy == 0) {
            T sum = 0;
            for (int jz = 0; jz < VALUE_PER_MOMENT; jz++)
                sum += *(T *) (tempvalues[i] + (jz << 1));
            *(T *) (moments + (blockMomentStart + i) * REC_SIZE_WORDS + KEY_SIZE_WORDS) = sum;
        }
    }
}

void computeCUDA(TypedCuboid &moments, const TypedCuboid &data) {

    const int halfmemorywords = (6UL * 1000 * 1000 * 1000) / sizeof(int);
    const int halfmemoryrecords = halfmemorywords / REC_SIZE_WORDS;

    int numDevices;
    hipGetDeviceCount(&numDevices);
    printf("Number of GPU detected = %d\n", numDevices);
    int **gpu_moments = new int *[numDevices];
    int **gpu_data = new int *[numDevices];
    auto startTime = high_resolution_clock::now();
    //Allocate memory on all devices
    for (int devId = 0; devId < numDevices; devId++) {
        checkCuda(hipSetDevice(devId));
        checkCuda(hipMalloc(gpu_moments + devId, halfmemoryrecords * REC_SIZE_WORDS * sizeof(int)));
        checkCuda(hipMalloc(gpu_data + devId, halfmemoryrecords * REC_SIZE_WORDS * sizeof(int)));
    }


    size_t stepCount = 0;
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    size_t momentsOffsets[numDevices];
    size_t numMoments[numDevices];

    for (int devId = 0; devId < numDevices; devId++) {
        checkCuda(hipSetDevice(devId));
        numMoments[devId] = std::min<size_t>(halfmemoryrecords, moments.numRows / numDevices);
        if (devId > 0)
            momentsOffsets[devId] = momentsOffsets[devId - 1] + numMoments[devId - 1];
        else
            momentsOffsets[0] = 0;

        printf("Device %d Moments from %lu to %lu (%lu moments) \n", devId, momentsOffsets[devId],
               momentsOffsets[devId] + numMoments[devId], numMoments[devId]);
        checkCuda(hipMemcpy(gpu_moments[devId], moments.ptr + momentsOffsets[devId] * REC_SIZE_WORDS,
                             numMoments[devId] * REC_SIZE_WORDS * sizeof(int),
                             hipMemcpyHostToDevice));
    }

    //For each block of data
    for (size_t dataoffset = 0; dataoffset < data.numRows; dataoffset += halfmemoryrecords) {

        //Copy the block of data to be computed to the devices. Same for all devices
        size_t numDataRemaining = std::min<size_t>(halfmemoryrecords, data.numRows - dataoffset);
        printf("NumDataRemaining = %lu \n", numDataRemaining);
        size_t numDataPerCent = numDataRemaining / 100 + 1;
        for (int devId = 0; devId < numDevices; devId++) {
            checkCuda(hipSetDevice(devId));
            checkCuda(hipMemcpy(gpu_data[devId], data.ptr + dataoffset * REC_SIZE_WORDS,
                                 numDataRemaining * REC_SIZE_WORDS * sizeof(int),
                                 hipMemcpyHostToDevice));
        }
        for (size_t start = 0; start < numDataRemaining; start += numDataPerCent) {
            for (int devId = 0; devId < numDevices; devId++) {
                checkCuda(hipSetDevice(devId));
                size_t thisdevicenumBlocks = numMoments[devId] / ENTRIES_IN_BLOCK + 1;
                computeKernel<<<thisdevicenumBlocks, NUMCORES>>>(gpu_moments[devId], gpu_data[devId],
                                                                  numMoments[devId], start, std::min(
                                start + numDataPerCent, numDataRemaining));
            }
            for (int devId = 0; devId < numDevices; devId++) {
                checkCuda(hipSetDevice(devId));
                checkCuda(hipDeviceSynchronize());
            }
            stepCount++;
            printf("Step %lu / 100 \n", stepCount);
        }
    }

    //Copy the result for the block of moments from respective devices
    for (int devId = 0; devId < numDevices; devId++) {
        checkCuda(hipSetDevice(devId));
        checkCuda(hipMemcpy(moments.ptr + momentsOffsets[devId] * REC_SIZE_WORDS, gpu_moments[devId],
                             numMoments[devId] * REC_SIZE_WORDS * sizeof(int),
                             hipMemcpyDeviceToHost));
    }


    auto endTime = high_resolution_clock::now();
    auto duration = duration_cast<seconds>(endTime - startTime).count();
    printf("Computation on GPU took %lu seconds \n", duration);
    double rate = moments.numRows * 0.0864 / duration;
    printf("Rate = %f M/day \n", rate);
    for (int devId = 0; devId < numDevices; devId++) {
        checkCuda(hipSetDevice(devId));
        checkCuda(hipFree(gpu_moments[devId]));
        checkCuda(hipFree(gpu_data[devId]));

    }
    delete[] gpu_moments;
    delete[] gpu_data;
}
